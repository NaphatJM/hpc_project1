#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>

#define THREADS_PER_BLOCK 128
#define MATRIX_SIZE 128

// CUDA Kernel for Selection Sort using Global Memory
__global__ void selectionSort_global(int* d_matrix, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n) {
        for (int j = 0; j < n - 1; j++) {
            int min_idx = j;
            for (int k = j + 1; k < n; k++) {
                int index1 = row * n + k;
                int index2 = row * n + min_idx;
                if (d_matrix[index1] < d_matrix[index2]) {
                    min_idx = k;
                }
            }
            if (min_idx != j) {
                int temp = d_matrix[row * n + j];
                d_matrix[row * n + j] = d_matrix[row * n + min_idx];
                d_matrix[row * n + min_idx] = temp;
            }
        }
    }
}

// CUDA Kernel for Selection Sort using Shared Memory
__global__ void selectionSort_shared(int* d_matrix, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int shared_row[MATRIX_SIZE];

    if (row < n) {
        int row_start = row * n;
        
        // Load row into shared memory
        for (int i = threadIdx.x; i < n; i += blockDim.x) {
            shared_row[i] = d_matrix[row_start + i];
        }
        __syncthreads();

        // Perform Selection Sort in Shared Memory
        for (int j = 0; j < n - 1; j++) {
            int min_idx = j;
            for (int k = j + 1; k < n; k++) {
                if (shared_row[k] < shared_row[min_idx]) {
                    min_idx = k;
                }
            }
            if (min_idx != j) {
                int temp = shared_row[j];
                shared_row[j] = shared_row[min_idx];
                shared_row[min_idx] = temp;
            }
        }
        __syncthreads();

        // Copy back sorted row to Global Memory
        for (int i = threadIdx.x; i < n; i += blockDim.x) {
            d_matrix[row_start + i] = shared_row[i];
        }
    }
}

// CUDA Kernel to find min - max values using Global Memory
__global__ void findMinMax_global(int* d_matrix, int* d_min, int* d_max, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n) {
        int minVal = d_matrix[row * n];
        int maxVal = d_matrix[row * n];

        for (int j = 1; j < n; j++) {
            int val = d_matrix[row * n + j];
            if (val < minVal) minVal = val;
            if (val > maxVal) maxVal = val;
        }
        
        d_min[row] = minVal;
        d_max[row] = maxVal;
    }
}

// CUDA Kernel to find min - max values using Shared Memory
__global__ void findMinMax_shared(int* d_matrix, int* d_min, int* d_max, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int shared_row[MATRIX_SIZE];
    
    if (row < n) {
        int row_start = row * n;
        
        for (int i = threadIdx.x; i < n; i += blockDim.x) {
            shared_row[i] = d_matrix[row_start + i];
        }
        __syncthreads();
        
        int minVal = shared_row[0];
        int maxVal = shared_row[0];
        for (int i = 1; i < n; i++) {
            if (shared_row[i] < minVal) minVal = shared_row[i];
            if (shared_row[i] > maxVal) maxVal = shared_row[i];
        }
        
        d_min[row] = minVal;
        d_max[row] = maxVal;
    }
}

int main() {
    std::ifstream file("random_matrix_128x128.csv");
    
    std::vector<int> host_matrix;
    int value;
    
    while (file >> value) {
        host_matrix.push_back(value);
        if (file.peek() == ',') file.ignore();
    }
    file.close();
    
    int size = MATRIX_SIZE * MATRIX_SIZE;
    int *d_matrix, *d_min, *d_max;

    hipMalloc(&d_matrix, size * sizeof(int));
    hipMalloc(&d_min, MATRIX_SIZE * sizeof(int));
    hipMalloc(&d_max, MATRIX_SIZE * sizeof(int));
    
    hipMemcpy(d_matrix, host_matrix.data(), size * sizeof(int), hipMemcpyHostToDevice);

//---------------------------------------------------------------------------------------------------------------------
    auto start = std::chrono::high_resolution_clock::now();
    selectionSort_global<<<MATRIX_SIZE, 1>>>(d_matrix, MATRIX_SIZE);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> global_sort_duration = end - start;
//---------------------------------------------------------------------------------------------------------------------

//---------------------------------------------------------------------------------------------------------------------
    start = std::chrono::high_resolution_clock::now();
    selectionSort_shared<<<MATRIX_SIZE, 1>>>(d_matrix, MATRIX_SIZE);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> shared_sort_duration = end - start;
//---------------------------------------------------------------------------------------------------------------------

//---------------------------------------------------------------------------------------------------------------------
    start = std::chrono::high_resolution_clock::now();
    findMinMax_global<<<MATRIX_SIZE, 1>>>(d_matrix, d_min, d_max, MATRIX_SIZE);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> global_minmax_duration = end - start;
//---------------------------------------------------------------------------------------------------------------------

//---------------------------------------------------------------------------------------------------------------------
    start = std::chrono::high_resolution_clock::now();
    findMinMax_shared<<<MATRIX_SIZE, 1>>>(d_matrix, d_min, d_max, MATRIX_SIZE);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> shared_minmax_duration = end - start;
//---------------------------------------------------------------------------------------------------------------------

    hipMemcpy(host_matrix.data(), d_matrix, size * sizeof(int), hipMemcpyDeviceToHost);
    std::vector<int> host_min(MATRIX_SIZE);
    std::vector<int> host_max(MATRIX_SIZE);
    hipMemcpy(host_min.data(), d_min, MATRIX_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(host_max.data(), d_max, MATRIX_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    std::ofstream sorted_file("sorted_selection_gpu.csv");
    for (int i = 0; i < MATRIX_SIZE; i++) {
        for (int j = 0; j < MATRIX_SIZE; j++) {
            sorted_file << host_matrix[i * MATRIX_SIZE + j];
            if (j < MATRIX_SIZE - 1) sorted_file << ",";
        }
        sorted_file << "\n";
    }
    sorted_file.close();

    std::cout << "Row-wise Min/Max values:\n";
    for (int i = 0; i < MATRIX_SIZE; i++) {
        std::cout << "Row " << (i + 1) << ": Min = " << host_min[i] << ", Max = " << host_max[i] << "\n";
    }

    std::cout << "--------------------------------" << std::endl;
    std::cout << "Global Memory Sorting: " << global_sort_duration.count() << " ms\n";
    std::cout << "Shared Memory Sorting: " << shared_sort_duration.count() << " ms\n";
    std::cout << "--------------------------------" << std::endl;
    std::cout << "Global Memory Min/Max: " << global_minmax_duration.count() << " ms\n";
    std::cout << "Shared Memory Min/Max: " << shared_minmax_duration.count() << " ms\n";

    hipFree(d_matrix);
    hipFree(d_min);
    hipFree(d_max);

    return 0;
}
